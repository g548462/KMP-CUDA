
#include <hip/hip_runtime.h>
#include<stdlib.h>
#include <stdio.h>
#include <string.h>
#include<time.h>
#define Size 10
#define patternSize 3
#define patternNum 20
#define ThreadNum 20 
#define BlockNum 1

__device__ void preKmp(char *x, int m, int kmpNext[])
{
	int i, j;
	i = 0;
	j = kmpNext[0] = -1;
	while(i < m)
	{
		while(j>-1 && x[i]!=x[j])
			j = kmpNext[j];
		i++;
		j++;
		if(x[i]==x[j])
			kmpNext[i] = kmpNext[j];
		else
			kmpNext[i] = j;

	}
}

/*******************************************

This variable   m:pattern.length    x:pattern
				n:array.length      y:array

*******************************************/
__device__ void KMP(char *x, int m, char *y, int n,int *answer,int id)
{
	int i, j, kmpNext[Size];

	preKmp(x,m,kmpNext);
	i = j = 0;
	while(j < n)
	{
		while(i>-1 && x[i]!=y[j])
		{
		  	i = kmpNext[i];
		}
		i++;
		j++;
		if(i >= m)
		{
			i = kmpNext[i];
			answer[id]=j-1;	
		}

	}
}

__global__ void kmp_kernel(char *array,char *pattern,int *answer)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;
  char *p;
  p=&pattern[id*(patternSize+1)];
  KMP(p,patternSize,array,Size,answer,id);
   
}

int main(int argc,char *argv[])
{
  int i=0,j=0,tmp,*answer,*d_answer;
  hipError_t r;
  char *array,*b,*pattern;
  char *d_array,*d_pattern;



  srand(time(0));
  array=(char*)malloc(sizeof(char)*Size);
  b=(char*)malloc(sizeof(char)*26);
  pattern=(char*)malloc(sizeof(char)*(patternSize+1)*patternNum);
  answer=(int*)malloc(sizeof(int)*patternNum);
  /************************************
  *   cudaMalloc
  ************************************/

  r=hipMalloc((void**)&d_array,sizeof(char)*Size);
  printf("hipMalloc d_array : %s\n",hipGetErrorString(r));
  r=hipMalloc((void**)&d_pattern,sizeof(char)*(patternSize+1)*patternNum);
  printf("hipMalloc d_pattern : %s\n",hipGetErrorString(r));
  r=hipMalloc((void**)&d_answer,sizeof(int)*patternNum);
  printf("hipMalloc d_answer : %s\n",hipGetErrorString(r));


  b="abcdefghijklmnopqrstuvwxyz";
  for(i=0;i<Size;i++)
	array[i]=b[rand()%26];

  for(i=0;i<patternNum;i++)
  {
	tmp=rand()%(Size-patternSize);
	for(j=0;j<patternSize+1;j++)
	{
	  if(j!=patternSize)
	  {
		pattern[i*(patternSize+1)+j]=array[tmp++];
		printf("%d   %c\n",i,array[tmp-1]);
	  }
	  else
	  {
		printf("===================== %d   \n",j);
		pattern[i*(patternSize+1)+j]='\0';
		printf("%c\n",pattern[i*patternSize+j]);
	  }
	}
  }
  for(i=0;i<patternNum;i++)
  {
	answer[i]=0;
  }


  r=hipMemcpy(d_array,array,sizeof(char)*Size,hipMemcpyHostToDevice);
  printf("Memcpy H->D d_array : %s\n",hipGetErrorString(r));
  
  r=hipMemcpy(d_pattern,pattern,sizeof(char)*(patternSize+1)*patternNum,hipMemcpyHostToDevice);
  printf("Memcpy H->D d_pattern : %s\n",hipGetErrorString(r));
  
  r=hipMemcpy(d_answer,answer,sizeof(int)*patternNum,hipMemcpyHostToDevice);
  printf("Memcpy H->D d_answer : %s\n",hipGetErrorString(r));
  
  kmp_kernel<<<BlockNum, ThreadNum>>>(d_array, d_pattern, d_answer);

  r=hipMemcpy(answer, d_answer, sizeof(int)*patternNum, hipMemcpyDeviceToHost);
  printf("Memcpy D->H answer : %s\n",hipGetErrorString(r));


  printf("Array:\n");
  printf("%s\n", array);
  for(i=0;i<(patternSize+1)*patternNum;i++)
	  printf("%c", pattern[i]);
  printf("\n\n");
  for(i=0;i<patternNum;i++)
	printf("%d, %d\n", i, answer[i]);
  //printf("array : %c\n",array[3]);
  //for(i=0;i<patternSize*patternNum;i++)
  	//printf("%s\n",pattern);
  
  //for(i=0;i<patternNum;i++)
	//printf("%d  %s\n",i,pattern[i]);

  //KMP<<<blockNum,threadNum>>>(b, strlen(b), array, strlen(array));

  
  return 0;
}
